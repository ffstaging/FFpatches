
#include <hip/hip_runtime.h>
/*
 * This file is part of FFmpeg.
 *
 * FFmpeg is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation; either
 * version 2.1 of the License, or (at your option) any later version.
 *
 * FFmpeg is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with FFmpeg; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA
 */


extern "C" {

__global__ void pad_planar_cuda(unsigned char* dst, int dst_pitch, int dst_w, int dst_h,
                                const unsigned char* src, int src_pitch, int src_w, int src_h,
                                int roi_x, int roi_y, unsigned char fill_val)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dst_w || y >= dst_h) {
        return;
    }

    if (x >= roi_x && x < (roi_x + src_w) && y >= roi_y && y < (roi_y + src_h)) {
        const int src_x = x - roi_x;
        const int src_y = y - roi_y;
        dst[y * dst_pitch + x] = src[src_y * src_pitch + src_x];
    } else {
        dst[y * dst_pitch + x] = fill_val;
    }
}

__global__ void pad_uv_cuda(uchar2* dst, int dst_pitch, int dst_w, int dst_h,
                            const uchar2* src, int src_pitch, int src_w, int src_h,
                            int roi_x, int roi_y, uchar2 fill_val)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dst_w || y >= dst_h) {
        return;
    }

    const int dst_pitch_uchar2 = dst_pitch / sizeof(uchar2);
    const int src_pitch_uchar2 = src_pitch / sizeof(uchar2);

    if (x >= roi_x && x < (roi_x + src_w) && y >= roi_y && y < (roi_y + src_h)) {
        const int src_x = x - roi_x;
        const int src_y = y - roi_y;
        dst[y * dst_pitch_uchar2 + x] = src[src_y * src_pitch_uchar2 + src_x];
    } else {
        dst[y * dst_pitch_uchar2 + x] = fill_val;
    }
}

}