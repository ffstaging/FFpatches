#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025, Faeez Kadiri < f1k2faeez at gmail dot com>
 *
 * This file is part of FFmpeg.
 *
 * FFmpeg is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation; either
 * version 2.1 of the License, or (at your option) any later version.
 *
 * FFmpeg is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with FFmpeg; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA
 */

#include "cuda/vector_helpers.cuh"

// Transpose direction constants (from transpose.h)
#define TRANSPOSE_CCLOCK_FLIP 0
#define TRANSPOSE_CLOCK       1
#define TRANSPOSE_CCLOCK      2
#define TRANSPOSE_CLOCK_FLIP  3
#define TRANSPOSE_REVERSAL    4
#define TRANSPOSE_HFLIP       5
#define TRANSPOSE_VFLIP       6

// FFmpeg passes pitch in bytes, CUDA uses potentially larger types
#define FIXED_PITCH(T) \
    (dst_pitch/sizeof(T))

#define DEFAULT_DST(n, T) \
    dst[n][yo*FIXED_PITCH(T)+xo]

// --- COORDINATE TRANSFORMATION FUNCTIONS ---

__device__ static inline void get_transpose_coords(int src_x, int src_y, int src_width, int src_height,
                                                  int *dst_x, int *dst_y, int dst_width, int dst_height, int dir)
{
    switch (dir) {
    case TRANSPOSE_CCLOCK_FLIP: // 90° CCW + vertical flip
        *dst_x = src_y;
        *dst_y = src_x;
        break;
    case TRANSPOSE_CLOCK: // 90° CW
        *dst_x = src_y;
        *dst_y = src_width - 1 - src_x;
        break;
    case TRANSPOSE_CCLOCK: // 90° CCW
        *dst_x = src_height - 1 - src_y;
        *dst_y = src_x;
        break;
    case TRANSPOSE_CLOCK_FLIP: // 90° CW + vertical flip
        *dst_x = src_height - 1 - src_y;
        *dst_y = src_width - 1 - src_x;
        break;
    case TRANSPOSE_REVERSAL: // 180° rotation
        *dst_x = src_width - 1 - src_x;
        *dst_y = src_height - 1 - src_y;
        break;
    case TRANSPOSE_HFLIP: // Horizontal flip
        *dst_x = src_width - 1 - src_x;
        *dst_y = src_y;
        break;
    case TRANSPOSE_VFLIP: // Vertical flip
        *dst_x = src_x;
        *dst_y = src_height - 1 - src_y;
        break;
    default:
        *dst_x = src_x;
        *dst_y = src_y;
        break;
    }
}

// --- TRANSPOSE KERNELS ---

#define TRANSPOSE_DEF(name, in_type, out_type) \
__device__ static inline void Transpose_##name##_impl( \
    hipTextureObject_t src_tex[4], out_type *dst[4], \
    int xo, int yo, int width, int height, int dst_pitch, \
    int dst_width, int dst_height, int src_width, int src_height, int dir) \
{ \
    int src_x, src_y; \
    get_transpose_coords(xo, yo, width, height, &src_x, &src_y, src_width, src_height, dir); \
    \
    in_type pixel = tex2D<in_type>(src_tex[0], src_x + 0.5f, src_y + 0.5f); \
    DEFAULT_DST(0, out_type) = pixel; \
}

#define TRANSPOSE_UV_DEF(name, in_type_uv, out_type_uv) \
__device__ static inline void Transpose_##name##_uv_impl( \
    hipTextureObject_t src_tex[4], out_type_uv *dst[4], \
    int xo, int yo, int width, int height, int dst_pitch, \
    int dst_width, int dst_height, int src_width, int src_height, int dir) \
{ \
    int src_x, src_y; \
    get_transpose_coords(xo, yo, width, height, &src_x, &src_y, src_width, src_height, dir); \
    \
    in_type_uv pixel_u = tex2D<in_type_uv>(src_tex[1], src_x + 0.5f, src_y + 0.5f); \
    in_type_uv pixel_v = tex2D<in_type_uv>(src_tex[2], src_x + 0.5f, src_y + 0.5f); \
    DEFAULT_DST(1, out_type_uv) = pixel_u; \
    DEFAULT_DST(2, out_type_uv) = pixel_v; \
}

#define TRANSPOSE_NV_UV_DEF(name, in_type_uv, out_type_uv) \
__device__ static inline void Transpose_##name##_uv_impl( \
    hipTextureObject_t src_tex[4], out_type_uv *dst[4], \
    int xo, int yo, int width, int height, int dst_pitch, \
    int dst_width, int dst_height, int src_width, int src_height, int dir) \
{ \
    int src_x, src_y; \
    get_transpose_coords(xo, yo, width, height, &src_x, &src_y, src_width, src_height, dir); \
    \
    in_type_uv pixel_uv = tex2D<in_type_uv>(src_tex[1], src_x + 0.5f, src_y + 0.5f); \
    DEFAULT_DST(1, out_type_uv) = pixel_uv; \
}


// Define transpose implementations for all formats
TRANSPOSE_DEF(yuv420p, uchar, uchar)
TRANSPOSE_UV_DEF(yuv420p, uchar, uchar)

TRANSPOSE_DEF(nv12, uchar, uchar)
TRANSPOSE_NV_UV_DEF(nv12, uchar2, uchar2)

TRANSPOSE_DEF(yuv444p, uchar, uchar)
TRANSPOSE_UV_DEF(yuv444p, uchar, uchar)

TRANSPOSE_DEF(p010le, ushort, ushort)
TRANSPOSE_NV_UV_DEF(p010le, ushort2, ushort2)

TRANSPOSE_DEF(p016le, ushort, ushort)
TRANSPOSE_NV_UV_DEF(p016le, ushort2, ushort2)

TRANSPOSE_DEF(yuv444p16le, ushort, ushort)
TRANSPOSE_UV_DEF(yuv444p16le, ushort, ushort)

TRANSPOSE_DEF(rgb0, uchar4, uchar4)
TRANSPOSE_DEF(bgr0, uchar4, uchar4)
TRANSPOSE_DEF(rgba, uchar4, uchar4)
TRANSPOSE_DEF(bgra, uchar4, uchar4)

// --- KERNEL ARGUMENT DEFINITIONS ---

#define TRANSPOSE_KERNEL_ARGS(T) \
    hipTextureObject_t src_tex_0, hipTextureObject_t src_tex_1, \
    hipTextureObject_t src_tex_2, hipTextureObject_t src_tex_3, \
    T *dst_0, T *dst_1, T *dst_2, T *dst_3, \
    int width, int height, int dst_pitch, \
    int dst_width, int dst_height, \
    int src_width, int src_height, int dir

#define TRANSPOSE_KERNEL_IMPL(func_impl, T) \
    hipTextureObject_t src_tex[4] = { src_tex_0, src_tex_1, src_tex_2, src_tex_3 }; \
    T *dst[4] = { dst_0, dst_1, dst_2, dst_3 }; \
    int xo = blockIdx.x * blockDim.x + threadIdx.x; \
    int yo = blockIdx.y * blockDim.y + threadIdx.y; \
    if (xo >= width || yo >= height) return; \
    \
    func_impl(src_tex, dst, xo, yo, width, height, dst_pitch, \
              dst_width, dst_height, src_width, src_height, dir);

extern "C" {

// --- TRANSPOSE KERNELS ---

#define TRANSPOSE_KERNEL(name, T) \
__global__ void Transpose_##name(TRANSPOSE_KERNEL_ARGS(T)) \
{ \
    TRANSPOSE_KERNEL_IMPL(Transpose_##name##_impl, T) \
}

#define TRANSPOSE_UV_KERNEL(name, T) \
__global__ void Transpose_##name##_uv(TRANSPOSE_KERNEL_ARGS(T)) \
{ \
    TRANSPOSE_KERNEL_IMPL(Transpose_##name##_uv_impl, T) \
}

// Transpose kernels for all formats
TRANSPOSE_KERNEL(yuv420p, uchar)
TRANSPOSE_UV_KERNEL(yuv420p, uchar)

TRANSPOSE_KERNEL(nv12, uchar)
TRANSPOSE_UV_KERNEL(nv12, uchar2)

TRANSPOSE_KERNEL(yuv444p, uchar)
TRANSPOSE_UV_KERNEL(yuv444p, uchar)

TRANSPOSE_KERNEL(p010le, ushort)
TRANSPOSE_UV_KERNEL(p010le, ushort2)

TRANSPOSE_KERNEL(p016le, ushort)
TRANSPOSE_UV_KERNEL(p016le, ushort2)

TRANSPOSE_KERNEL(yuv444p16le, ushort)
TRANSPOSE_UV_KERNEL(yuv444p16le, ushort)

TRANSPOSE_KERNEL(rgb0, uchar4)
TRANSPOSE_KERNEL(bgr0, uchar4)
TRANSPOSE_KERNEL(rgba, uchar4)
TRANSPOSE_KERNEL(bgra, uchar4)

// For RGB formats, UV kernels are not needed, but we provide empty implementations
// to maintain consistency with the function loading logic

#define EMPTY_UV_KERNEL(name, T) \
__global__ void Transpose_##name##_uv(TRANSPOSE_KERNEL_ARGS(T)) { } \

EMPTY_UV_KERNEL(rgb0, uchar)
EMPTY_UV_KERNEL(bgr0, uchar)
EMPTY_UV_KERNEL(rgba, uchar)
EMPTY_UV_KERNEL(bgra, uchar)

}
