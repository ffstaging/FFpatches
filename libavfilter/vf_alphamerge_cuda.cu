
#include <hip/hip_runtime.h>
/*
 * This file is part of FFmpeg.
 *
 * FFmpeg is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation; either
 * version 2.1 of the License, or (at your option) any later version.
 *
 * FFmpeg is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with FFmpeg; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA
 */


template <typename T>
__device__ void alphamerge_impl(T *dst, int dst_pitch,
                                const T *src, int src_pitch,
                                int width, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        dst[y * dst_pitch + x] = src[y * src_pitch + x];
    }
}

extern "C" {
    __global__ void alphamerge_planar(unsigned char* main_alpha_plane,
                                      int main_alpha_linesize,
                                      const unsigned char* alpha_mask_luma_plane,
                                      int alpha_mask_luma_linesize,
                                      int width, int height)
    {
        alphamerge_impl<unsigned char>(main_alpha_plane, main_alpha_linesize,
                                       alpha_mask_luma_plane, alpha_mask_luma_linesize,
                                       width, height);
    }
}